#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/AccumulateType.h>
#include <ATen/native/Pool.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/CUDAApplyUtils.cuh>
#include <ATen/cuda/detail/TensorInfo.cuh>
#include <ATen/cuda/detail/IndexUtils.cuh>
#include <ATen/cuda/detail/KernelUtils.h>
#include <THC/THCNumerics.cuh>
#include <c10/macros/Macros.h>


namespace at {
namespace native {
namespace {

__device__ inline int min(int a, int b) {
  return a <= b ? a : b;
}

// kernels borrowed from Caffe
template <typename scalar_t, typename accscalar_t>
__global__ void MaxPoolForward(const int nthreads, const scalar_t* bottom_data,
    const int num, const int depth, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    const int in_stride_c, const int in_stride_h, const int in_stride_w,
    scalar_t* top_data, int64_t* top_mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % depth;
    int n = index / pooled_width / pooled_height / depth;
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    int hend = min(hstart + (kernel_h - 1) * dilation_h + 1, height);
    int wend = min(wstart + (kernel_w - 1) * dilation_w + 1, width);
    while(hstart < 0)
      hstart += dilation_h;
    while(wstart < 0)
      wstart += dilation_w;
    accscalar_t maxval = at::numeric_limits<accscalar_t>::lower_bound(); // -Infinity
    int maxidx = hstart * in_stride_h + wstart * in_stride_w;
    bottom_data += (n * depth * height * width + c * in_stride_c);
    for (int h = hstart; h < hend; h += dilation_h) {
      for (int w = wstart; w < wend; w += dilation_w) {
        scalar_t val = bottom_data[h * in_stride_h + w * in_stride_w];
        if ((ScalarConvert<scalar_t, accscalar_t>::to(val) > maxval) || THCNumerics<scalar_t>::isnan(val)) {
          maxidx = h * in_stride_h + w * in_stride_w;
          maxval = ScalarConvert<scalar_t, accscalar_t>::to(val);
        }
      }
    }
    top_data[index] = ScalarConvert<scalar_t, accscalar_t>::to(maxval);
    top_mask[index] = maxidx;
  }
}

static const int BACKWARD_THREADS = 256;

template <typename scalar_t, typename accscalar_t>
#if defined (__HIP_PLATFORM_HCC__)
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 4)
#else
C10_LAUNCH_BOUNDS_2(BACKWARD_THREADS, 8)
#endif
__global__ void MaxPoolBackward(const int nthreads, const scalar_t* top_diff,
    const int64_t* top_mask, const int num, const int depth,
    const int height, const int width, const int pooled_height,
    const int pooled_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    const int dilation_h, const int dilation_w,
    const int out_stride_c, const int out_stride_h, const int out_stride_w,
    const int in_stride_c, const int in_stride_h, const int in_stride_w,
    scalar_t* bottom_diff) {
    CUDA_KERNEL_LOOP(index, height*width) {
    int h = index/width;
    int w = index - h * width;
//get some templating performance benefits without actually templating
    int phstart, phend, pwstart, pwend;
    if (stride_h == 1) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1))  + 1;
       phend = min((h + pad_h)  + 1, pooled_height);
    } else if (stride_h == 2) {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / 2  + 1;
       phend = min((h + pad_h) / 2  + 1, pooled_height);
    } else {
       phstart =
        (h + pad_h < ((kernel_h - 1) * dilation_h + 1)) ? 0 : (h + pad_h - ((kernel_h - 1) * dilation_h + 1)) / stride_h  + 1;
       phend = min((h + pad_h) / stride_h  + 1, pooled_height);
    }
    if (stride_w == 1) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) + 1;
        pwend = min((w + pad_w) + 1, pooled_width);
    } else if (stride_w == 2) {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / 2 + 1;
        pwend = min((w + pad_w) / 2 + 1, pooled_width);
    } else {
        pwstart =
        (w + pad_w < ((kernel_w - 1) * dilation_w + 1)) ? 0 : (w + pad_w - ((kernel_w - 1) * dilation_w + 1)) / stride_w + 1;
        pwend = min((w + pad_w) / stride_w + 1, pooled_width);
    }
    for (int n = blockIdx.y; n < num; n += gridDim.y)
       for (int c = blockIdx.z; c < depth; c+= gridDim.z) {

        accscalar_t gradient = accscalar_t(0);
        int offset = (n * depth * pooled_height * pooled_width + c * out_stride_c);
        top_diff += offset;
        top_mask += offset;
//get some templating performance benefits without actually templating
        if ((phstart + 1 != phend) || (pwstart + 1 != pwend)) {
        for (int ph = phstart; ph < phend; ++ph) {
          for (int pw = pwstart; pw < pwend; ++pw) {
            if (top_mask[ph * out_stride_h + pw * out_stride_w] == index) {
              gradient += ScalarConvert<scalar_t, accscalar_t>::to(top_diff[ph * out_stride_h + pw * out_stride_w]);
            }
          }
        }
        } else {
            if (top_mask[phstart * out_stride_h + pwstart * out_stride_w] == index) {
              gradient += ScalarConvert<scalar_t, accscalar_t>::to(top_diff[phstart * out_stride_h + pwstart * out_stride_w]);
            }
        }
        bottom_diff[n*depth*height*width + c * in_stride_c + h * in_stride_h + w * in_stride_w] = ScalarConvert<accscalar_t, scalar_t>::to(gradient);
      }
  }
}

void max_pool2d_with_indices_out_cuda_template(
           Tensor& output,
           Tensor& indices,
           const Tensor& input_,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg output_arg{ output, "output", 1 };
  TensorArg indices_arg{ indices, "indices", 2 };
  TensorArg input_arg{ input_, "input_", 3 };

  checkAllSameGPU("max_pool2d_with_indices_out_cuda",
                  {output_arg, indices_arg, input_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK((kernel_size.size() == 1 || kernel_size.size() == 2) &&
              (stride.empty() || stride.size() == 2) &&
              (padding.size() == 1 || padding.size() == 2) &&
              (dilation.size() == 1 || dilation.size() == 2),
    "max_pool2d_with_indices: internal error: all IntArrayRef sizes must be 2");

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW : safe_downcast<int, int64_t>(stride[1]);

  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  const auto memory_format = input_.suggest_memory_format();

  const int64_t nbatch = input_.ndimension() == 4 ? input_.size(-4) : 1;
  const int64_t size3 = input_.size(-3); // nInputPlane or inputHeight
  const int64_t size2 = input_.size(-2); // inputHeight or inputWidth
  const int64_t size1 = input_.size(-1); // inputWidth or nInputPlane

  int64_t stride3;
  int64_t stride2;
  int64_t stride1;
  int64_t inputWidth;
  int64_t inputHeight;
  if (memory_format == MemoryFormat::ChannelsLast)  {
    stride1 = 1; // channels stride
    stride2 = size1; // width stride = channels size
    stride3 = size2 * size1; // height stride = channels * width
    inputWidth = size2;
    inputHeight = size3;
  } else {
    stride1 = size2 * size1; // channel stride = height * width
    stride2 = size1; // height stride = width
    stride3 = 1; // width stride
    inputWidth = size1;
    inputHeight = size2;
  }

  const int64_t outputWidth = pooling_output_shape<int64_t>(inputWidth, kW, padW, dW, dilationW, ceil_mode);
  const int64_t outputHeight = pooling_output_shape<int64_t>(inputHeight, kH, padH, dH, dilationH, ceil_mode);

  pool2d_shape_check(
    input_,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    size3,
    size2, size1,
    outputHeight, outputWidth);

  Tensor input = input_.contiguous(memory_format);

  output.resize_({nbatch, size3, size2, size1});
  indices.resize_({nbatch, size3, size2, size1});

  const int count = safe_downcast<int, int64_t>(output.numel());
  const int num_threads = std::min(at::cuda::getCurrentDeviceProperties()->maxThreadsPerBlock,
                                   BACKWARD_THREADS);

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(),
    "max_pool2d_with_indices_out_cuda_frame",
    [&] {
      using accscalar_t = acc_type<scalar_t, true>;

      scalar_t *output_data = output.data<scalar_t>();
      scalar_t *input_data = input.data<scalar_t>();
      int64_t *indices_data = indices.data<int64_t>();

      MaxPoolForward<scalar_t, scalar_t>
        <<<cuda::ATenCeilDiv(count, num_threads), num_threads, 0, at::cuda::getCurrentCUDAStream()>>>(
          count, input_data,
          nbatch, size3, size2, size1, outputHeight, outputWidth,
          kH, kW, dH, dW, padH, padW, dilationH, dilationW,
          stride3, stride2, stride1,
          output_data, indices_data); }
  );

  TORCH_CHECK(hipGetLastError() == hipSuccess,
     "max_pool2d_with_indices_out_cuda_frame failed with error code ",
     hipGetLastError());

  if(input.ndimension() == 3) {
    output.resize_({size3, size2, size1});
  }
}

void max_pool2d_with_indices_backward_out_cuda_template(
           Tensor& gradInput,
           const Tensor& gradOutput_,
           const Tensor& input_,
           const Tensor& indices,
           IntArrayRef kernel_size,
           IntArrayRef stride,
           IntArrayRef padding,
           IntArrayRef dilation,
           bool ceil_mode)
{
  TensorArg gradInput_arg{ gradInput, "gradInput", 1 };
  TensorArg gradOutput_arg{ gradOutput_, "gradOutput_", 2 };
  TensorArg input_arg{ input_, "input_", 3 };
  TensorArg indices_arg{ indices, "indices", 4 };

  checkAllSameGPU("max_pool2d_with_indices_out_cuda",
                  {gradInput_arg, gradOutput_arg, input_arg, indices_arg});

  // #20866, #22032: Guarantee this for the official C++ API?
  TORCH_CHECK((kernel_size.size() == 1 || kernel_size.size() == 2) &&
              (stride.empty() || stride.size() == 2) &&
              (padding.size() == 1 || padding.size() == 2) &&
              (dilation.size() == 1 || dilation.size() == 2),
    "max_pool2d_with_indices: internal error: all IntArrayRef sizes must be 2");

  TORCH_CHECK((input_.ndimension() == 3 || input_.ndimension() == 4),
    "non-empty 3D or 4D (batch mode) tensor expected for input");

  const int kH = safe_downcast<int, int64_t>(kernel_size[0]);
  const int kW = kernel_size.size() == 1 ? kH : safe_downcast<int, int64_t>(kernel_size[1]);

  const int dH = stride.empty() ? kH : safe_downcast<int, int64_t>(stride[0]);
  const int dW = stride.empty() ? kW : safe_downcast<int, int64_t>(stride[1]);

  const int padH = safe_downcast<int, int64_t>(padding[0]);
  const int padW = padding.size() == 1 ? padH : safe_downcast<int, int64_t>(padding[1]);

  const int dilationH = safe_downcast<int, int64_t>(dilation[0]);
  const int dilationW = dilation.size() == 1 ? dilationH : safe_downcast<int, int64_t>(dilation[1]);

  const auto memory_format = input_.suggest_memory_format();

  const Tensor input = input_.contiguous(memory_format);

  const int64_t nbatch = input_.ndimension() == 4 ? input_.size(-4) : 1;
  const int64_t size3 = input_.size(-3); // nInputPlane or inputHeight
  const int64_t size2 = input_.size(-2); // inputHeight or inputWidth
  const int64_t size1 = input_.size(-1); // inputWidth or nInputPlane

  int64_t i_stride3;
  int64_t i_stride2;
  int64_t i_stride1;
  int64_t outputHeight;
  int64_t outputWidth;
  int64_t o_stride3;
  int64_t o_stride2;
  int64_t o_stride1;
  if (memory_format == MemoryFormat::ChannelsLast)  {
    i_stride1 = 1; // channels stride
    i_stride2 = size1; // width stride = channels size
    i_stride3 = size2 * size1; // height stride = channels * width
    outputHeight = pooling_output_shape<int64_t>(size3, kH, padH, dH, dilationH, ceil_mode);
    outputWidth = pooling_output_shape<int64_t>(size2, kW, padW, dW, dilationW, ceil_mode);
    o_stride1 = 1;
    o_stride2 = outputWidth;
    o_stride3 = outputHeight * outputWidth;
  } else {
    i_stride1 = size2 * size1; // channel stride = height * width
    i_stride2 = size1; // height stride = width
    i_stride3 = 1; // width stride
    outputHeight = pooling_output_shape<int64_t>(size2, kH, padH, dH, dilationH, ceil_mode);
    outputWidth = pooling_output_shape<int64_t>(size1, kW, padW, dW, dilationW, ceil_mode);
    o_stride1 = outputHeight * outputWidth;
    o_stride2 = outputWidth;
    o_stride3 = 1;
  }

  max_pool2d_backward_shape_check(
    input_,
    gradOutput_,
    indices,
    nbatch,
    kH, kW, dH, dW, padH, padW, dilationH, dilationW,
    size3,
    size2, size1,
    outputHeight, outputWidth,
    /*cuda=*/ true);

  const Tensor gradOutput = gradOutput_.contiguous(memory_format);
  gradInput.resize_as_(input);

  int64_t count = input.numel();
  dim3 grid;
  int imgcount = size2 * size1;
  const int blocks = (imgcount + BACKWARD_THREADS - 1) / BACKWARD_THREADS;
  grid.x = blocks;
  grid.y = nbatch;
  grid.z = size3;
  uint64_t maxGridY = at::cuda::getCurrentDeviceProperties()->maxGridSize[1];
  uint64_t maxGridZ = at::cuda::getCurrentDeviceProperties()->maxGridSize[2];
  if (maxGridY < grid.y) grid.y = maxGridY;
  if (maxGridZ < grid.z) grid.z = maxGridZ;

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(input.scalar_type(),
    "max_pool2d_with_indices_out_cuda_frame",
    [&] {
      using accscalar_t = acc_type<scalar_t, true>;

      scalar_t *gradOutput_data = gradOutput.data<scalar_t>();
      scalar_t *gradInput_data = gradInput.data<scalar_t>();
      int64_t *indices_data = indices.data<int64_t>();

      MaxPoolBackward<scalar_t, accscalar_t>
        <<<grid, BACKWARD_THREADS, 0, at::cuda::getCurrentCUDAStream()>>>(
          count,
          gradOutput_data,
          indices_data,
          nbatch,
          size3, size2, size1, outputHeight, outputWidth,
          kH, kW, dH, dW, padH, padW, dilationH, dilationW,
          o_stride3, o_stride2, o_stride1,
          i_stride3, i_stride2, i_stride1,
          gradInput_data);
    }
  );

  TORCH_CHECK(hipGetLastError() == hipSuccess,
    "fractional_max_pool2d_backward_out_cuda failed with error code ",
    hipGetLastError());
}

} // namespace

std::tuple<Tensor&, Tensor&> max_pool2d_with_indices_out_cuda(
  Tensor& output,
  Tensor& indices,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  max_pool2d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return std::tuple<Tensor&, Tensor&>(output, indices);
}

std::tuple<Tensor, Tensor> max_pool2d_with_indices_cuda(
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode)
{
  Tensor output = at::empty({0}, input.options());
  Tensor indices = at::empty({0}, input.options().dtype(kLong));
  max_pool2d_with_indices_out_cuda_template(
    output,
    indices,
    input,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return std::tuple<Tensor, Tensor>(output, indices);
}

Tensor& max_pool2d_with_indices_backward_out_cuda(
  Tensor& gradInput,
  const Tensor& gradOutput_,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  max_pool2d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

Tensor max_pool2d_with_indices_backward_cuda(
  const Tensor& gradOutput_,
  const Tensor& input,
  IntArrayRef kernel_size,
  IntArrayRef stride,
  IntArrayRef padding,
  IntArrayRef dilation,
  bool ceil_mode,
  const Tensor& indices)
{
  auto gradInput = at::zeros_like(input);
  max_pool2d_with_indices_backward_out_cuda_template(
    gradInput,
    gradOutput_,
    input,
    indices,
    kernel_size,
    stride,
    padding,
    dilation,
    ceil_mode);
  return gradInput;
}

} // at::native
} // at
